#include "hip/hip_runtime.h"
#include "unary.cuh"

template <class T>
static __global__ void op_abs(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = fabsf(x[i]);
}

template <class T>
static __global__ void op_sgn(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)(x[i] > (T)0.f ? 1.f : ((x[i] < (T)0.f ? -1.f : 0.f)));
}

template <class T>
static __global__ void op_neg(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = -x[i];
}

template <class T>
static __global__ void op_step(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = x[i] > (T)0.0f;
}

template <class T>
static __global__ void op_gelu(const T * x, T * dst, const int k) {
    const T GELU_COEF_A    = 0.044715f;
    const T SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    T xi = x[i];
    dst[i] = (T)0.5f*xi*((T)1.0f + (T)tanhf(SQRT_2_OVER_PI*xi*((T)1.0f + GELU_COEF_A*xi*xi)));
}

template <class T>
static __global__ void gelu_erf_f32(const T * x, T * dst, const int k) {
    const T SQRT_2_INV = 0.70710678118654752440084436210484f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = (T) 0.5f * x[i] * ((T)1.0f + (T)erff(SQRT_2_INV * x[i]));
}

// static __global__ void gelu_quick_f32(const float * x, float * dst, int k) {
    // const float GELU_QUICK_COEF = -1.702f;

template <class T>
static __global__ void op_gelu_quick(const T * x, T * dst, int k) {
    const T GELU_QUICK_COEF = -1.702f;
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = x[i] * ((T)1.0f / ((T)1.0f + (T)expf(GELU_QUICK_COEF * x[i])));
}

template <class T>
static __global__ void op_silu(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / ((T)1.0f + (T)expf(-x[i]));
}

template <class T>
static __global__ void op_silu_back(
        const T * grad, const T * xf, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    const T xfi = xf[i];
    const T s = (T)1.0f / ((T)1.0f + (T)expf(-xfi));
    dst[i] = grad[i] * s * ((T)1.0f + xfi * ((T)1.0f - s));
}

template <class T>
static __global__ void fused_mul_silu_f32(const T * x, const T * y, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * y[i] / ((T)1.0f + (T)expf(-x[i]));
}

static __global__ void multi_add_f32(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, const char * src0, char * dst) {

    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;
    int64_t k = ne0*ne1;
    if (i >= k) {
        return;
    }
    int i1 = i / ne0;
    int i0 = i % ne0;
    float * result = (float *)(dst + i1*nb1);
    const float * s = (const float *)(src0 + i1*nb01) + i0;
    if (nused == 1) {
        result[i0] = s[0];
    } else {
        float sum = s[0] + s[ne0];
        for (int j = 2; j < nused; ++j) sum += s[j*ne0];
        result[i0] = sum;
    }
}

// template <class T>
// static __global__ void multi_add_f32(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, const char * src0, T * dst) {
    // const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;
    // int64_t k = ne0*ne1;
    // if (i >= k) {
        // return;
    // }
    // int i1 = i / ne0;
    // int i0 = i % ne0;
    // T * result = (T *)(dst + i1*nb1);
    // const float * s = (const float *)(src0 + i1*nb01) + i0;
    // if (nused == 1) {
        // result[i0] = s[0];
    // } else {
        // T sum = s[0] + s[ne0];
        // for (int j = 2; j < nused; ++j) sum += s[j*ne0];
        // result[i0] = sum;
    // }
// }

template <class T>
static __global__ void fused_mul_relu_f32(const T * x, const T * y, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0) * y[i];
}

template <class T>
static __global__ void fused_mul_gelu_f32(const T * x, const T * y, T * dst, const int k) {
    constexpr T GELU_COEF_A    = 0.044715f;
    constexpr T SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    T xi = x[i];
    dst[i] = (T)0.5f*xi*y[i]*((T)1.0f + (T)tanhf(SQRT_2_OVER_PI*xi*((T)1.0f + (T)GELU_COEF_A*xi*xi)));
}

// static __global__ void tanh_f32(const float * x, float * dst, int k) {

template <class T>
static __global__ void op_tanh(const T * x, T * dst, int k) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = tanhf(x[i]);
}

template <class T>
static __global__ void op_relu(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0);
}

template <class T>
static __global__ void op_sigmoid(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = (T)1.0f / ((T)1.0f + (T)expf(-x[i]));
}

template <class T>
static __global__ void op_hardsigmoid(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fminf(1.0f, fmaxf(0.0f, (x[i] + (T)3.0f) / (T)6.0f));
}

template <class T>
static __global__ void op_hardswish(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * (T)fminf(1.0f, fmaxf(0.0f, (x[i] + (T)3.0f) / (T)6.0f));
}

template <class T>
static __global__ void op_exp(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = expf(x[i]);
}

template <class T>
static __global__ void op_leaky_relu(const T * x, T * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = (T)fmaxf(x[i], 0) + (T)fminf(x[i], 0.0f) * (T)negative_slope;
}

template <class T>
static __global__ void op_sqr(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * x[i];
}

template <class T>
static __global__ void op_sqrt(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sqrtf(x[i]);
}

template <class T>
static __global__ void op_sin(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sinf(x[i]);
}

template <class T>
static __global__ void op_cos(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = cosf(x[i]);
}

template <class T>
static __global__ void op_log(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = logf(x[i]);
}

template <class T>
static void abs_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    op_abs<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sgn_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    op_sgn<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void neg_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    op_neg<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void step_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_STEP_BLOCK_SIZE - 1) / CUDA_STEP_BLOCK_SIZE;
    op_step<<<num_blocks, CUDA_STEP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void gelu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    op_gelu<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void gelu_erf_f32_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_erf_f32<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

// static void gelu_quick_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {

template <class T>
static void gelu_quick_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    op_gelu_quick<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void silu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    op_silu<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void silu_back_cuda(const T * grad, const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BACK_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    op_silu_back<<<num_blocks, CUDA_SILU_BACK_BLOCK_SIZE, 0, stream>>>(grad, x, dst, k);
}

template <class T>
static void fused_mul_silu_f32_cuda(const T * x, const T * y, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    fused_mul_silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

template <class T>
static void fused_mul_relu_f32_cuda(const T * x, const T * y, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    fused_mul_relu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

template <class T>
static void fused_mul_gelu_f32_cuda(const T * x, const T * y, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    fused_mul_gelu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

// static void tanh_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {

template <class T>
static void tanh_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_TANH_BLOCK_SIZE - 1) / CUDA_TANH_BLOCK_SIZE;
    op_tanh<<<num_blocks, CUDA_TANH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void relu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    op_relu<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sigmoid_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIGMOID_BLOCK_SIZE - 1) / CUDA_SIGMOID_BLOCK_SIZE;
    op_sigmoid<<<num_blocks, CUDA_SIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void hardsigmoid_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSIGMOID_BLOCK_SIZE - 1) / CUDA_HARDSIGMOID_BLOCK_SIZE;
    op_hardsigmoid<<<num_blocks, CUDA_HARDSIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void hardswish_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSWISH_BLOCK_SIZE - 1) / CUDA_HARDSWISH_BLOCK_SIZE;
    op_hardswish<<<num_blocks, CUDA_HARDSWISH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void exp_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_EXP_BLOCK_SIZE - 1) / CUDA_EXP_BLOCK_SIZE;
    op_exp<<<num_blocks, CUDA_EXP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void leaky_relu_cuda(const T * x, T * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    op_leaky_relu<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

template <class T>
static void sqr_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQR_BLOCK_SIZE - 1) / CUDA_SQR_BLOCK_SIZE;
    op_sqr<<<num_blocks, CUDA_SQR_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sqrt_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQRT_BLOCK_SIZE - 1) / CUDA_SQRT_BLOCK_SIZE;
    op_sqrt<<<num_blocks, CUDA_SQRT_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sin_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIN_BLOCK_SIZE - 1) / CUDA_SIN_BLOCK_SIZE;
    op_sin<<<num_blocks, CUDA_SIN_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void cos_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_COS_BLOCK_SIZE - 1) / CUDA_COS_BLOCK_SIZE;
    op_cos<<<num_blocks, CUDA_COS_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void log_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_COS_BLOCK_SIZE - 1) / CUDA_COS_BLOCK_SIZE;
    op_log<<<num_blocks, CUDA_COS_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

void ggml_cuda_op_abs(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        abs_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        abs_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sgn(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sgn_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sgn_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_neg(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        neg_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        neg_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_step(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        step_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        step_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void multi_add_f32_cuda(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, const char * src0, char * dst, hipStream_t stream) {
    int64_t k = ne0 * ne1;
    const int num_blocks = (k + CUDA_MULTI_ADD_BLOCK_SIZE - 1) / CUDA_MULTI_ADD_BLOCK_SIZE;
    multi_add_f32<<<num_blocks, CUDA_MULTI_ADD_BLOCK_SIZE, 0, stream>>>(nused, ne0, ne1, nb1, nb01, src0, dst);
}

void ggml_cuda_op_multi_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->ne[2] == 1 && dst->ne[3] == 1);
    GGML_ASSERT(dst->nb[0] == sizeof(float));
    int nused = dst->op_params[0];
    GGML_ASSERT(nused >= 1);
    const char * src0 = (const char *)dst->src[0]->data;
    hipStream_t stream = ctx.stream();
    multi_add_f32_cuda(nused, dst->ne[0], dst->ne[1], dst->nb[1], dst->src[0]->nb[1], (char *)src0, (char *)dst->data, stream);
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        gelu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        gelu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_silu_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // input from forward pass
    const ggml_tensor * src1 = dst->src[1]; // grads of forward pass output

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_back_cuda((const half *)src0_d, (const half *)src1_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_back_cuda((const float*)src0_d, (const float*)src1_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_fused_mul_unary(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_are_same_shape(src0, dst));
    GGML_ASSERT(ggml_are_same_shape(src0, src1));

    hipStream_t stream = ctx.stream();
    ggml_unary_op op = (ggml_unary_op)dst->op_params[0];

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;
    float * dst_d = (float *)dst->data;

    switch (op) {
        case GGML_UNARY_OP_SILU: fused_mul_silu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
        case GGML_UNARY_OP_RELU: fused_mul_relu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
        case GGML_UNARY_OP_GELU: fused_mul_gelu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
        default: GGML_ASSERT(false);
    }
}

void ggml_cuda_op_gelu_erf(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    gelu_erf_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        gelu_quick_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        gelu_quick_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        tanh_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        tanh_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sigmoid_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sigmoid_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        hardsigmoid_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        hardsigmoid_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        hardswish_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        hardswish_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_exp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        exp_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        exp_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    if (src0->type == GGML_TYPE_F16) {
        leaky_relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), negative_slope, stream);
    } else {
        leaky_relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), negative_slope, stream);
    }
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sqr_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sqr_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sqrt_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sqrt_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sin(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sin_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sin_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_cos(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        cos_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        cos_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_log(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        log_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        log_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}
